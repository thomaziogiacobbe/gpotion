/*-----------------------------------------------------------
 ** gaussian.cu -- The program is to solve a linear system Ax = b
 **   by using Gaussian Elimination. The algorithm on page 101
 **   ("Foundations of Parallel Programming") is used.  
 **   The sequential version is gaussian.c.  This parallel 
 **   implementation converts three independent for() loops 
 **   into three Fans.  Use the data file ge_3.dat to verify 
 **   the correction of the output. 
 **
 ** Written by Andreas Kura, 02/15/95
 ** Modified by Chong-wei Xu, 04/20/95
 ** Modified by Chris Gregg for CUDA, 07/20/2009
 ** Modified by André Du Bois, 12/19/2023: clenead up code that I was not using
 **-----------------------------------------------------------
 */
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <string.h>
#include <math.h>

#define MAXBLOCKSIZE 512
#define BLOCK_SIZE_XY 4


int Size;
float *a, *b, *finalVec;
float *m;

void InitPerRun();
void ForwardSub();
void BackSub();
__global__ void Fan1(float *m, float *a, int Size, int t);
__global__ void Fan2(float *m, float *a, float *b,int Size, int j1, int t);

void PrintMat(float *ary, int nrow, int ncolumn);
void PrintAry(float *ary, int ary_size);
void checkCUDAError(const char *msg);


// create both matrix and right hand side, Ke Wang 2013/08/12 11:51:06
void
create_matrix(float *m, int size){
  int i,j;
  float lamda = -0.01;
  float coe[2*size-1];
  float coe_i =0.0;

  for (i=0; i < size; i++)
    {
      coe_i = 10*exp(lamda*i); 
      j=size-1+i;     
      coe[j]=coe_i;
      j=size-1-i;     
      coe[j]=coe_i;
    }


  for (i=0; i < size; i++) {
      for (j=0; j < size; j++) {
	m[i*size+j]=coe[size-1-i+j];
      }
  }


}


int main(int argc, char *argv[])
{
 // printf("WG size of kernel 1 = %d, WG size of kernel 2= %d X %d\n", MAXBLOCKSIZE, BLOCK_SIZE_XY, BLOCK_SIZE_XY);
    
	     if (argc != 2)
		 {
			printf("Please inform size!\n");
			exit(0);
		 }
   	
          Size = atoi(argv[1]);
	
	      a = (float *) malloc(Size * Size * sizeof(float));
	      create_matrix(a, Size);

	      b = (float *) malloc(Size * sizeof(float));
	      for (int j =0; j< Size; j++)
	    	b[j]=1.0;

	      m = (float *) malloc(Size * Size * sizeof(float));

    InitPerRun();
    //begin timing
    
    // run kernels
	float time;
    hipEvent_t start, stop;   
     hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    ForwardSub();
    

    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    printf("CUDA\t%d\t%3.1f\n", Size,time);


    BackSub();
    
    free(m);
    free(a);
    free(b);

}
/*------------------------------------------------------
 ** PrintDeviceProperties
 **-----------------------------------------------------
 */
 
 
/*------------------------------------------------------
 ** InitProblemOnce -- Initialize all of matrices and
 ** vectors by opening a data file specified by the user.
 **
 ** We used dynamic array *a, *b, and *m to allocate
 ** the memory storages.
 **------------------------------------------------------
 */

/*------------------------------------------------------
 ** InitPerRun() -- Initialize the contents of the
 ** multipier matrix **m
 **------------------------------------------------------
 */
void InitPerRun() 
{
	int i;
	for (i=0; i<Size*Size; i++)
			*(m+i) = 0.0;
}

/*-------------------------------------------------------
 ** Fan1() -- Calculate multiplier matrix
 ** Pay attention to the index.  Index i give the range
 ** which starts from 0 to range-1.  The real values of
 ** the index should be adjust and related with the value
 ** of t which is defined on the ForwardSub().
 **-------------------------------------------------------
 */
__global__ void Fan1(float *m_cuda, float *a_cuda, int Size, int t)
{   
	//if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) printf(".");
	//printf("blockIDx.x:%d,threadIdx.x:%d,Size:%d,t:%d,Size-1-t:%d\n",blockIdx.x,threadIdx.x,Size,t,Size-1-t);

	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	*(m_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) = *(a_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) / *(a_cuda+Size*t+t);
}

/*-------------------------------------------------------
 ** Fan2() -- Modify the matrix A into LUD
 **-------------------------------------------------------
 */ 

__global__ void Fan2(float *m_cuda, float *a_cuda, float *b_cuda,int Size, int j1, int t)
{
	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	if(threadIdx.y + blockIdx.y * blockDim.y >= Size-t) return;
	
	int xidx = blockIdx.x * blockDim.x + threadIdx.x;
	int yidx = blockIdx.y * blockDim.y + threadIdx.y;
	//printf("blockIdx.x:%d,threadIdx.x:%d,blockIdx.y:%d,threadIdx.y:%d,blockDim.x:%d,blockDim.y:%d\n",blockIdx.x,threadIdx.x,blockIdx.y,threadIdx.y,blockDim.x,blockDim.y);
	
	a_cuda[Size*(xidx+1+t)+(yidx+t)] -= m_cuda[Size*(xidx+1+t)+t] * a_cuda[Size*t+(yidx+t)];
	//a_cuda[xidx+1+t][yidx+t] -= m_cuda[xidx+1+t][t] * a_cuda[t][yidx+t];
	if(yidx == 0){
		//printf("blockIdx.x:%d,threadIdx.x:%d,blockIdx.y:%d,threadIdx.y:%d,blockDim.x:%d,blockDim.y:%d\n",blockIdx.x,threadIdx.x,blockIdx.y,threadIdx.y,blockDim.x,blockDim.y);
		//printf("xidx:%d,yidx:%d\n",xidx,yidx);
		b_cuda[xidx+1+t] -= m_cuda[Size*(xidx+1+t)+(yidx+t)] * b_cuda[t];
	}
}

/*------------------------------------------------------
 ** ForwardSub() -- Forward substitution of Gaussian
 ** elimination.
 **------------------------------------------------------
 */
void ForwardSub()
{
	int t;
    float *m_cuda,*a_cuda,*b_cuda;
	
	// allocate memory on GPU
	hipMalloc((void **) &m_cuda, Size * Size * sizeof(float));
	 
	hipMalloc((void **) &a_cuda, Size * Size * sizeof(float));
	
	hipMalloc((void **) &b_cuda, Size * sizeof(float));	

	// copy memory to GPU
	hipMemcpy(m_cuda, m, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(a_cuda, a, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(b_cuda, b, Size * sizeof(float),hipMemcpyHostToDevice );
	
	int block_size,grid_size;
	
	block_size = MAXBLOCKSIZE;
	grid_size = (Size/block_size) + (!(Size%block_size)? 0:1);
	//printf("1d grid size: %d\n",grid_size);


	dim3 dimBlock(block_size);
	dim3 dimGrid(grid_size);
	//dim3 dimGrid( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );
	
	int blockSize2d, gridSize2d;
	blockSize2d = BLOCK_SIZE_XY;
	gridSize2d = (Size/blockSize2d) + (!(Size%blockSize2d?0:1)); 
	
	dim3 dimBlockXY(blockSize2d,blockSize2d);
	dim3 dimGridXY(gridSize2d,gridSize2d);
	for (t=0; t<(Size-1); t++) {
		Fan1<<<dimGrid,dimBlock>>>(m_cuda,a_cuda,Size,t);
		hipDeviceSynchronize();
		Fan2<<<dimGridXY,dimBlockXY>>>(m_cuda,a_cuda,b_cuda,Size,Size-t,t);
		hipDeviceSynchronize();
		checkCUDAError("Fan2");
	}
	// end timing kernels

	// copy memory back to CPU
	hipMemcpy(m, m_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(a, a_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(b, b_cuda, Size * sizeof(float),hipMemcpyDeviceToHost );
	hipFree(m_cuda);
	hipFree(a_cuda);
	hipFree(b_cuda);
}

/*------------------------------------------------------
 ** BackSub() -- Backward substitution
 **------------------------------------------------------
 */

void BackSub()
{
	// create a new vector to hold the final answer
	finalVec = (float *) malloc(Size * sizeof(float));
	// solve "bottom up"
	int i,j;
	for(i=0;i<Size;i++){
		finalVec[Size-i-1]=b[Size-i-1];
		for(j=0;j<i;j++)
		{
			finalVec[Size-i-1]-=*(a+Size*(Size-i-1)+(Size-j-1)) * finalVec[Size-j-1];
		}
		finalVec[Size-i-1]=finalVec[Size-i-1]/ *(a+Size*(Size-i-1)+(Size-i-1));
	}
}


/*------------------------------------------------------
 ** PrintMat() -- Print the contents of the matrix
 **------------------------------------------------------
 */
void PrintMat(float *ary, int nrow, int ncol)
{
	int i, j;
	
	for (i=0; i<nrow; i++) {
		for (j=0; j<ncol; j++) {
			printf("%8.2f ", *(ary+Size*i+j));
		}
		printf("\n");
	}
	printf("\n");
}


/*------------------------------------------------------
 ** PrintAry() -- Print the contents of the array (vector)
 **------------------------------------------------------
 */
void PrintAry(float *ary, int ary_size)
{
	int i;
	for (i=0; i<ary_size; i++) {
		printf("%.2f ", ary[i]);
	}
	printf("\n\n");
}
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

